#include "hip/hip_runtime.h"

#include <thread>
#include <vector>
#include <future>
#include <string>
#include <memory>
#include <random>

#include "iPic3D.h"
#include "VCtopology3D.h"
#include "outputPrepare.h"
#include "threadPool.hpp"

#include "dataAnalysis.cuh"
#include "dataAnalysisConfig.cuh"
#include "GMM/cudaGMM.cuh"
#include "particleArraySoACUDA.cuh"
#include "velocityHistogram.cuh"



namespace dataAnalysis
{

using namespace iPic3D;
using velocitySoA = particleArraySoA::particleArraySoACUDA<cudaCommonType, 0, 3>;
using namespace std;
using namespace cudaGMMWeight;

class dataAnalysisPipelineImpl {
using weightType = cudaTypeSingle;
private:
    int ns;
    int deviceOnNode;
    // pointers to objects in KCode
    hipStream_t* streams;
    particleArrayCUDA** pclsArrayHostPtr = nullptr;

    std::future<int> analysisFuture;

    ThreadPool* DAthreadPool = nullptr;
    velocitySoA* velocitySoACUDA = nullptr;
    // histogram
    string HistogramSubDomainOutputPath;
    velocityHistogram::velocityHistogram* velocityHistogram = nullptr;

    // GMM
    string GMMSubDomainOutputPath;
    cudaGMMWeight::GMM<cudaCommonType, GMM_DATA_DIM, weightType>* gmmArray = nullptr;

    vector<array<vector<GMMResult<cudaCommonType, GMM_DATA_DIM>>, 3>> gmmResults;

public:

    dataAnalysisPipelineImpl(c_Solver& KCode) {
        ns = KCode.ns;
        deviceOnNode = KCode.cudaDeviceOnNode;
        streams = KCode.streams;
        pclsArrayHostPtr = KCode.pclsArrayHostPtr;

        DAthreadPool = new ThreadPool(4);

        if constexpr (VELOCITY_HISTOGRAM_ENABLE) { // velocity histogram
            velocitySoACUDA = new velocitySoA();

            HistogramSubDomainOutputPath = HISTOGRAM_OUTPUT_DIR + "subDomain" + std::to_string(KCode.myrank) + "/";
            velocityHistogram = new velocityHistogram::velocityHistogram(VELOCITY_HISTOGRAM_RES * VELOCITY_HISTOGRAM_RES);

            if constexpr (GMM_ENABLE) { // GMM
                GMMSubDomainOutputPath = GMM_OUTPUT_DIR + "subDomain" + std::to_string(KCode.myrank) + "/";
                gmmArray = new cudaGMMWeight::GMM<cudaCommonType, GMM_DATA_DIM, weightType>[3];

                if constexpr (GMM_OUTPUT) gmmResults.resize(ns);

            }
        }
    }

    void startAnalysis(int cycle);

    int checkAnalysis();

    int waitForAnalysis();


    ~dataAnalysisPipelineImpl() {

        if constexpr (GMM_OUTPUT) {
            std::string uvw[3] = {"/uv", "/vw", "/uw"};

            // output the GMM results
            int i=0, j=0;
            for (auto& speciesResArray : gmmResults) {
                for (auto& plane : speciesResArray) {
                    string planePath = GMMSubDomainOutputPath + "species" + std::to_string(i) + uvw[j] + ".json";
                    GMMResult<cudaCommonType, GMM_DATA_DIM>::outputResultArray(plane, planePath, uvw[j]); 
                    j++;  
                }
                i++;
            }
        }

        if (DAthreadPool != nullptr) delete DAthreadPool;
        if (velocitySoACUDA != nullptr) delete velocitySoACUDA;
        if (velocityHistogram != nullptr) delete velocityHistogram;
        if (gmmArray != nullptr) delete[] gmmArray;
    }

private:

    int analysisEntre(int cycle);

    int GMMAnalysisSpecies(const int cycle, const int species, const std::string outputPath);
    
};



/**
 * @brief analysis function for each species, uv, uw, vw
 * @details It launches 3 threads for uv uw vw analysis in parallel
 * 
 */
int dataAnalysisPipelineImpl::GMMAnalysisSpecies(const int cycle, const int species, const std::string outputPath){

    using weightType = cudaTypeSingle;

    std::future<int> future[3];

    auto GMMLambda = [=](int i) mutable {

        using namespace cudaGMMWeight;

        // GMM config
        // set the random number generator
        std::random_device rd;  // True random seed
        std::mt19937 gen(rd()); // Mersenne Twister PRN
        const cudaCommonType maxVelocity = species == 0 || species == 2 ? MAX_VELOCITY_HIST_E : MAX_VELOCITY_HIST_I;

        // it is assumed that GMM_DATA_DIM == 2 and thta the velocity range is homogenues in all dimensions
        const cudaCommonType maxVelocityArray[GMM_DATA_DIM] = {maxVelocity,maxVelocity};

        std::uniform_real_distribution<cudaCommonType> distR(1e-8, maxVelocity);
        std::uniform_real_distribution<cudaCommonType> distTheta(0, 2*M_PI);
        

        cudaErrChk(hipSetDevice(deviceOnNode));

        cudaCommonType weightVector[NUM_COMPONENT_GMM];
        cudaCommonType meanVector[NUM_COMPONENT_GMM * GMM_DATA_DIM];
        cudaCommonType coVarianceMatrix[NUM_COMPONENT_GMM * GMM_DATA_DIM * GMM_DATA_DIM ];

        const cudaCommonType uth = species == 0 || species == 2 ? 0.045 : 0.0126;
        const cudaCommonType vth = species == 0 || species == 2 ? 0.045 : 0.0126;
        const cudaCommonType wth = species == 0 || species == 2 ? 0.045 : 0.0126;
        
        cudaCommonType var1 = 0.01;
        cudaCommonType var2 = 0.01; 
        
        if (i==0)
        {
            var1 = uth;
            var2 = vth;
        }
        else if(i==1)
        {
            var1 = uth;
            var2 = wth;
        }
        else if(i==2)
        {
            var1 = vth;
            var2 = wth;
        }
        
        
        cudaCommonType normalization = 1.0;
        // normalize initial parameters if NORMALIZE_DATA_FOR_GMM==true
        if constexpr(NORMALIZE_DATA_FOR_GMM)
        {
            normalization = maxVelocity;
        }
        for(int j = 0; j < NUM_COMPONENT_GMM; j++){
            weightVector[j] = 1.0/NUM_COMPONENT_GMM;
            cudaCommonType radius = distR(gen);
            cudaCommonType theta = distTheta(gen);
            meanVector[j * 2] =  radius*cos(theta)/normalization;
            meanVector[j * 2 + 1] = radius*sin(theta)/normalization;
            coVarianceMatrix[j * 4] = var1/(normalization*normalization);
            coVarianceMatrix[j * 4 + 1] = 0.0;
            coVarianceMatrix[j * 4 + 2] = 0.0;
            coVarianceMatrix[j * 4 + 3] = var2/(normalization*normalization);
        }

        GMMParam_t<cudaCommonType> GMMParam = {
            .numComponents = NUM_COMPONENT_GMM,
            .maxIteration = MAX_ITERATION_GMM,
            .threshold = THRESHOLD_CONVERGENCE_GMM,

            .weightInit = weightVector,
            .meanInit = meanVector,
            .coVarianceInit = coVarianceMatrix
        };

        // data
        GMMDataMultiDim<cudaCommonType, GMM_DATA_DIM, weightType> GMMData
            (VELOCITY_HISTOGRAM_RES*VELOCITY_HISTOGRAM_RES, 
                velocityHistogram->getHistogramScaleMark(i), 
                velocityHistogram->getVelocityHistogramCUDAArray(i), 
                {maxVelocityArray[0], maxVelocityArray[1]});

        cudaErrChk(hipHostRegister(&GMMData, sizeof(GMMData), hipHostRegisterDefault));
        
        // generate exact output file path
        std::string uvw[3] = {"/uv_", "/vw_", "/uw_"};
        auto fileOutputPath = outputPath + uvw[i] + std::to_string(cycle) + ".json";

        auto& gmm = gmmArray[i];
        gmm.config(&GMMParam, &GMMData);
        auto convergStep = gmm.initGMM(); // the exact output file name
        int ret = 0;
        if constexpr (GMM_OUTPUT) {
            ret = gmm.outputGMM(convergStep, fileOutputPath); // immediate output

            // results vector
            gmmResults[species][i].push_back(gmm.getGMMResult(cycle, convergStep));
        }

        cudaErrChk(hipHostUnregister(&GMMData));
        
        return ret;
    };

    for(int i = 0; i < 3; i++){
        // launch 3 async threads for uv, uw, vw
        future[i] = DAthreadPool->enqueue(GMMLambda, i); 
    }

    for(int i = 0; i < 3; i++){
        future[i].wait();
    }

    return 0;
}

/**
 * @brief analysis function, called by startAnalysis
 * @details procesures in this function should be executed in sequence, the order of the analysis should be defined here
 *          But the procedures can launch other threads to do the analysis
 *          Also this function is a friend function of c_Solver, resources in the c_Slover should be dispatched here
 */
int dataAnalysisPipelineImpl::analysisEntre(int cycle){
    cudaErrChk(hipSetDevice(deviceOnNode));

    // species by species to save VRAM
    for(int i = 0; i < ns; i++){
        if constexpr (VELOCITY_HISTOGRAM_ENABLE) {
            // to SoA
            velocitySoACUDA->updateFromAoS(pclsArrayHostPtr[i], streams[i]);

            // histogram
            auto histogramSpeciesOutputPath = HistogramSubDomainOutputPath + "species" + std::to_string(i) + "/";
            velocityHistogram->init(velocitySoACUDA, cycle, i, streams[i]);
            if constexpr (HISTOGRAM_OUTPUT)
            velocityHistogram->writeToFile(histogramSpeciesOutputPath, streams[i]); // TODO
            else cudaErrChk(hipStreamSynchronize(streams[i]));

            if constexpr (GMM_ENABLE) { // GMM
                auto GMMSpeciesOutputPath = GMMSubDomainOutputPath + "species" + std::to_string(i) + "/";
                GMMAnalysisSpecies(cycle, i, GMMSpeciesOutputPath);
            }
        }
    }

    return 0;
}


/**
 * @brief start all the analysis registered here
 */
void dataAnalysisPipelineImpl::startAnalysis(int cycle){

    if(DATA_ANALYSIS_EVERY_CYCLE == 0 || (cycle % DATA_ANALYSIS_EVERY_CYCLE != 0)){
        analysisFuture = std::future<int>();
    } else {
        analysisFuture = DAthreadPool->enqueue(&dataAnalysisPipelineImpl::analysisEntre, this, cycle); 

        if(analysisFuture.valid() == false){
            throw std::runtime_error("[!]Error: Can not start data analysis");
        }
    }

}

/**
 * @brief check if the analysis is done, non-blocking
 * 
 * @return 0 if the analysis is done, 1 if it is not done
 */
int dataAnalysisPipelineImpl::checkAnalysis(){

    if(analysisFuture.valid() == false){
        return 0;
    }

    if(analysisFuture.wait_for(std::chrono::seconds(0)) == std::future_status::ready){
        return 0;
    }else{
        return 1;
    }

    return 0;
}

/**
 * @brief wait for the analysis to be done, blocking
 */
int dataAnalysisPipelineImpl::waitForAnalysis(){

    if(analysisFuture.valid() == false){
        return 0;
    }

    analysisFuture.wait();

    return 0;
}


/**
 * @brief create output directory for the data analysis, controlled by dataAnalysisConfig.cuh
 */
void dataAnalysisPipeline::createOutputDirectory(int myrank, int ns, VirtualTopology3D* vct){ // output path for data analysis
    if constexpr (DATA_ANALYSIS_ENABLED == false){
        return;
    }

    // VCT mapping for this subdomain
    auto writeVctMapping = [&](const std::string& filePath) {
        std::ofstream vctMapping(filePath);
        if(vctMapping.is_open()){
        vctMapping << "Cartesian rank: " << vct->getCartesian_rank() << std::endl;
        vctMapping << "Number of processes: " << vct->getNprocs() << std::endl;
        vctMapping << "XLEN: " << vct->getXLEN() << std::endl;
        vctMapping << "YLEN: " << vct->getYLEN() << std::endl;
        vctMapping << "ZLEN: " << vct->getZLEN() << std::endl;
        vctMapping << "X: " << vct->getCoordinates(0) << std::endl;
        vctMapping << "Y: " << vct->getCoordinates(1) << std::endl;
        vctMapping << "Z: " << vct->getCoordinates(2) << std::endl;
        vctMapping << "PERIODICX: " << vct->getPERIODICX() << std::endl;
        vctMapping << "PERIODICY: " << vct->getPERIODICY() << std::endl;
        vctMapping << "PERIODICZ: " << vct->getPERIODICZ() << std::endl;

        vctMapping << "Neighbor X left: " << vct->getXleft_neighbor() << std::endl;
        vctMapping << "Neighbor X right: " << vct->getXright_neighbor() << std::endl;
        vctMapping << "Neighbor Y left: " << vct->getYleft_neighbor() << std::endl;
        vctMapping << "Neighbor Y right: " << vct->getYright_neighbor() << std::endl;
        vctMapping << "Neighbor Z left: " << vct->getZleft_neighbor() << std::endl;
        vctMapping << "Neighbor Z right: " << vct->getZright_neighbor() << std::endl;

        vctMapping.close();
        } else {
        throw std::runtime_error("[!]Error: Can not create VCT mapping for velocity GMM species");
        }
    };

    if constexpr (VELOCITY_HISTOGRAM_ENABLE && HISTOGRAM_OUTPUT) {
        auto histogramSubDomainOutputPath = HISTOGRAM_OUTPUT_DIR + "subDomain" + std::to_string(myrank) + "/";
        for(int i = 0; i < ns; i++){
            auto histogramSpeciesOutputPath = histogramSubDomainOutputPath + "species" + std::to_string(i);
            if(0 != checkOutputFolder(histogramSpeciesOutputPath)){
            throw std::runtime_error("[!]Error: Can not create output folder for velocity histogram species");
            }
        }
        writeVctMapping(histogramSubDomainOutputPath + "vctMapping.txt");
    }

    if constexpr (GMM_ENABLE && GMM_OUTPUT) {
        auto GMMSubDomainOutputPath = GMM_OUTPUT_DIR + "subDomain" + std::to_string(myrank) + "/";
        for(int i = 0; i < ns; i++){
            auto GMMSpeciesOutputPath = GMMSubDomainOutputPath + "species" + std::to_string(i) + "/";
            if(0 != checkOutputFolder(GMMSpeciesOutputPath)){
            throw std::runtime_error("[!]Error: Can not create output folder for velocity GMM species");
            }
        }
        writeVctMapping(GMMSubDomainOutputPath + "vctMapping.txt");
    }

}



dataAnalysisPipeline::dataAnalysisPipeline(iPic3D::c_Solver& KCode) {
    if constexpr (DATA_ANALYSIS_ENABLED == false){
        impl = nullptr;
        return;
    }
    impl = std::make_unique<dataAnalysisPipelineImpl>(std::ref(KCode));
}

void dataAnalysisPipeline::startAnalysis(int cycle) {
    if constexpr (DATA_ANALYSIS_ENABLED == false){
        return;
    }
    impl->startAnalysis(cycle);
}

int dataAnalysisPipeline::checkAnalysis() {
    if constexpr (DATA_ANALYSIS_ENABLED == false){
        return 0;
    }
    return impl->checkAnalysis();
}

int dataAnalysisPipeline::waitForAnalysis() {
    if constexpr (DATA_ANALYSIS_ENABLED == false){
        return 0;
    }
    return impl->waitForAnalysis();
}

dataAnalysisPipeline::~dataAnalysisPipeline() {
    
}
    
} // namespace dataAnalysis







